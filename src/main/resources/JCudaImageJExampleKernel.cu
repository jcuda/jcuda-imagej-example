
#include <hip/hip_runtime.h>
extern "C"
__global__ void invert(uchar4* data, int w, int h)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    if (x < w && y < h)
    {
        int index = y*w+x;
        uchar4 pixel = data[index];
        pixel.x = 255 - pixel.x;
        pixel.y = 255 - pixel.y;
        pixel.z = 255 - pixel.z;
        pixel.w = 255 - pixel.w;
        data[index] = pixel;
    }
}
